#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include "hip/hip_runtime.h"
#include ""
#include <array>
#include <assert.h>
#include <atomic>
#include <chrono>
#include <iostream>
#include <limits>
#include <memory>
#include <stdint.h>
#include <vector>

#include "testdata.h"
#include "atomic.cuh"
#include "prefix.cuh"
#include "util.h"

typedef float enumf;
typedef float enumi;

struct PerfCounter
{
  unsigned long long *counter;

  __device__ __host__ inline PerfCounter(unsigned long long *target) : counter(target) {}

  __device__ __host__ inline void count() { aggregated_atomic_inc(counter);
  }

  __device__ __host__ inline void perf_count(unsigned int c, unsigned long long time)
  {
    if (thread_id() == 0)
    atomic_add(&counter[c], time);
  }
};

template <unsigned int levels, unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
struct SubtreeEnumerationBuffer;

__device__ __host__ inline enumi next_coeff(enumi coeff, const enumf center)
{
  const enumf rounded_center = round(center);
  coeff                      = 2 * rounded_center - coeff;
  if (center >= rounded_center)
  {
    return coeff + static_cast<int>(coeff >= rounded_center);
  }
  else
  {
    return coeff - static_cast<int>(coeff <= rounded_center);
  }
}

struct Matrix
{
  const enumf *ptr;
  unsigned int ld;

  __device__ __host__ inline Matrix(const enumf *ptr, unsigned int ld) : ptr(ptr), ld(ld) {}

  __device__ __host__ inline Matrix() : ptr(nullptr), ld(0) {}

  __device__ __host__ inline enumf at(unsigned int row, unsigned int col) const
  {
    return ptr[row * ld + col];
  }

  __device__ __host__ inline Matrix block(unsigned int start_row, unsigned int start_col) const {
    return Matrix(&ptr[start_col + start_row * ld], ld);
  }
};

template <unsigned int maxdim> struct CudaEnumeration
{
  enumi x[maxdim];
  enumf partdist[maxdim];
  // ! different to base enumeration of fplll, the second index is shifted !
  // _[i][j] contains inner product of i-th orthogonalized basis vector with B * (0, ..., 0, x[j +
  // 1], ... x[n])
  enumf center_partsums[maxdim][maxdim];
  enumf center[maxdim];
  const uint32_t *radius_squared_location;

  // row-major
  Matrix mu;
  const enumf *rdiag;

  template <int kk, typename Callback>
  __device__ __host__ bool enumerate_recursive(Callback &, unsigned int &max_paths, PerfCounter& counter);

  template <int kk> __device__ __host__ bool is_enumeration_done() const;

  __device__ __host__ enumf get_radius_squared();

  template <unsigned int levels, unsigned int dimensions_per_level,
            unsigned int max_nodes_per_level>
  friend struct SubtreeEnumerationBuffer;
};

template <unsigned int maxdim>
__device__ __host__ inline enumf CudaEnumeration<maxdim>::get_radius_squared()
{
  return int_to_float_order_preserving_bijection(*radius_squared_location);
}

template <unsigned int maxdim>
template <int kk>
__device__ __host__ inline bool CudaEnumeration<maxdim>::is_enumeration_done() const
{
  if constexpr (kk >= 0)
  {
    return isnan(x[kk]);
  }
  else
  {
    return false;
  }
}

/**
 * Searches the subtree of height kk + 1 using as root the values stored in this object. The
 * reference max_paths contains an integer that gives the maximal count of tree paths to search
 * (including tree paths that lead to nodes with too great partdist and are therefore cut). After
 * this is exceeded, the tree search is aborted but can be resumed later by calling
 * enumerate_recursive on this object. The function returns whether the subtree was completely
 * searched.
 */
template <unsigned int maxdim>
template <int kk, typename Callback>
__device__ __host__ inline bool
CudaEnumeration<maxdim>::enumerate_recursive(Callback &callback, unsigned int &max_paths,
                                             PerfCounter& counter)
{
  static_assert(kk < static_cast<int>(maxdim),
                "Tree level count must be <= maximal enumeration dimension count");
  assert(max_paths >= 1);
  if constexpr (kk >= 0)
  {
    enumf alphak     = x[kk] - center[kk];
    enumf newdist = partdist[kk] + alphak * alphak * rdiag[kk];

    if (!(newdist <= get_radius_squared()))
    {
      x[kk] = NAN;
      return true;
    }

    if constexpr (kk == 0)
    {
      callback(x, newdist);
    }
    else
    {
      partdist[kk - 1] = newdist;

      for (int j = 0; j < kk; ++j)
      {
        center_partsums[j][kk - 1] = center_partsums[j][kk] - x[kk] * mu.at(j, kk);
      }
      assert(!isnan(center_partsums[kk - 1][kk - 1]));

      center[kk - 1]           = center_partsums[kk - 1][kk - 1];
      if (isnan(x[kk - 1]))
      {
        x[kk - 1] = round(center[kk - 1]);
      }
    }

    while (true)
    {
      counter.count();
      bool is_done = enumerate_recursive<kk - 1, Callback>(callback, max_paths, counter);
      if (!is_done)
      {
        return false;
      }

      x[kk] = next_coeff(x[kk], center[kk]);

      enumf alphak2  = x[kk] - center[kk];
      enumf newdist2 = partdist[kk] + alphak2 * alphak2 * rdiag[kk];
      assert(!isnan(newdist2));

      if (max_paths == 1)
      {
        return false;
      }
      --max_paths;

      if (!(newdist2 <= get_radius_squared()))
      {
        x[kk] = NAN;
        return true;
      }

      if constexpr (kk == 0)
      {
        callback(x, newdist2);
      }
      else
      {
        partdist[kk - 1] = newdist2;

        for (int j = 0; j < kk; ++j)
        {
          center_partsums[j][kk - 1] = center_partsums[j][kk] - x[kk] * mu.at(j, kk);
        }
        assert(!isnan(center_partsums[kk - 1][kk - 1]));

        center[kk - 1] = center_partsums[kk - 1][kk - 1];
        x[kk - 1]      = round(center[kk - 1]);
      }
    }
  }
  return true;
}

template <unsigned int levels, unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
struct SubtreeEnumerationBuffer
{
private:
  // shape [levels, dimensions_per_level, max_nodes_per_level]
  enumi * enumeration_x;
  // shape [levels, dimensions_per_level, max_nodes_per_level]
  enumi * coefficients;
  // shape [levels, dimensions, max_nodes_per_level], of subtree root
  enumf * center_partsum;
  // shape [levels, max_nodes_per_level], of subtree root
  enumf * partdist;
  // shape [levels, max_nodes_per_level]
  unsigned int * parent_indices;

  // shape [levels]
  unsigned int *open_node_count;

  constexpr static unsigned int dimensions = levels * dimensions_per_level;

  constexpr static unsigned int enumeration_x_size_in_bytes =
      sizeof(enumi) * levels * dimensions_per_level * max_nodes_per_level;

  constexpr static unsigned int coefficient_size_in_bytes =
      sizeof(enumi) * levels * dimensions_per_level * max_nodes_per_level;

  constexpr static unsigned int center_partsum_size_in_bytes =
      sizeof(enumf) * levels * dimensions * max_nodes_per_level;

  constexpr static unsigned int partdist_size_in_bytes =
      sizeof(enumf) * levels * max_nodes_per_level;

  constexpr static unsigned int parent_indices_size_in_bytes =
      sizeof(unsigned int) * levels * max_nodes_per_level;

  constexpr static unsigned int open_node_count_size_in_bytes =
      sizeof(unsigned int) * levels;

public:
  __device__ __host__ inline SubtreeEnumerationBuffer(unsigned char *memory)
      : center_partsum(reinterpret_cast<enumf *>(memory)),
        partdist(reinterpret_cast<enumf *>(memory + center_partsum_size_in_bytes)),
        enumeration_x(reinterpret_cast<enumi *>(memory + center_partsum_size_in_bytes +
                                                partdist_size_in_bytes)),
        coefficients(reinterpret_cast<enumi *>(memory + center_partsum_size_in_bytes +
                                               partdist_size_in_bytes +
                                               enumeration_x_size_in_bytes)),
        parent_indices(reinterpret_cast<unsigned int *>(
            memory + center_partsum_size_in_bytes + partdist_size_in_bytes +
            enumeration_x_size_in_bytes + coefficient_size_in_bytes)),
        open_node_count(reinterpret_cast<unsigned int *>(
            memory + center_partsum_size_in_bytes + partdist_size_in_bytes +
            enumeration_x_size_in_bytes + coefficient_size_in_bytes + parent_indices_size_in_bytes))
  {
  }

  template <typename CG> __device__ __host__ inline void init(CG &cooperative_group)
  {
    if (cooperative_group.thread_rank() == 0)
    {
      for (unsigned int i = 0; i < levels; ++i)
      {
        open_node_count[i] = 0;
      }
    }
  }

  constexpr static size_t memory_size_in_bytes =
      enumeration_x_size_in_bytes + coefficient_size_in_bytes +
      center_partsum_size_in_bytes + partdist_size_in_bytes +
      parent_indices_size_in_bytes + open_node_count_size_in_bytes;

  static_assert(memory_size_in_bytes < std::numeric_limits<unsigned int>::max(),
                "Requires more memory than indexable with unsigned int");

  __device__ __host__ inline CudaEnumeration<dimensions_per_level>
  get_enumeration(unsigned int tree_level, unsigned int index,
                  Matrix mu_block, const enumf *rdiag,
                  const uint32_t *radius_squared_location)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    CudaEnumeration<dimensions_per_level> result;

    const unsigned int offset_kk = (levels - tree_level - 1) * dimensions_per_level;

    result.mu                      = mu_block;
    result.rdiag                   = &rdiag[offset_kk];
    result.radius_squared_location = radius_squared_location;

    for (unsigned int i = 0; i < dimensions_per_level; ++i)
    {
      result.x[i] = enumeration_x[tree_level * dimensions_per_level * max_nodes_per_level +
                                  i * max_nodes_per_level + index];

      const enumf center_partsum_i = center_partsum[tree_level * dimensions * max_nodes_per_level +
                                                    (offset_kk + i) * max_nodes_per_level + index];
      assert(!isnan(center_partsum_i));
      result.center_partsums[i][dimensions_per_level - 1] = center_partsum_i;
    }
    result.center[dimensions_per_level - 1] =
        center_partsum[tree_level * dimensions * max_nodes_per_level +
                       (offset_kk + dimensions_per_level - 1) * max_nodes_per_level + index];
    result.partdist[dimensions_per_level - 1] = partdist[tree_level * max_nodes_per_level + index];
    return result;
  }

  __device__ __host__ inline void set_enumeration(unsigned int tree_level, unsigned int index,
                                           const CudaEnumeration<dimensions_per_level> &value)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    const unsigned int offset_kk = (levels - tree_level - 1) * dimensions_per_level;

    for (unsigned int i = 0; i < dimensions_per_level; ++i)
    {
      enumeration_x[tree_level * dimensions_per_level * max_nodes_per_level +
                    i * max_nodes_per_level + index] = value.x[i];

      const enumf old_parent_center_partsum =
          center_partsum[tree_level * dimensions * max_nodes_per_level +
                         (offset_kk + i) * max_nodes_per_level + index];
      const enumf new_parent_center_partsum = value.center_partsums[i][dimensions_per_level - 1];
      assert(!isnan(old_parent_center_partsum));
      assert(!isnan(new_parent_center_partsum));
      assert(old_parent_center_partsum == new_parent_center_partsum);
    }
    assert(center_partsum[tree_level * dimensions * max_nodes_per_level +
                          (offset_kk + dimensions_per_level - 1) * max_nodes_per_level + index] ==
           value.center[dimensions_per_level - 1]);
    assert(partdist[tree_level * max_nodes_per_level + index] ==
           value.partdist[dimensions_per_level - 1]);
  }

  __device__ __host__ inline void init_subtree(unsigned int tree_level, unsigned int index,
                                        enumf parent_partdist, enumf center)
  {
    for (unsigned int i = 0; i < dimensions_per_level; ++i)
    {
      enumeration_x[tree_level * dimensions_per_level * max_nodes_per_level +
                    i * max_nodes_per_level + index] = NAN;
    }
    partdist[tree_level * max_nodes_per_level + index]                      = parent_partdist;
    enumeration_x[tree_level * dimensions_per_level * max_nodes_per_level +
                  (dimensions_per_level - 1) * max_nodes_per_level + index] = round(center);
  }

  __device__ __host__ inline void set_center_partsum(unsigned int tree_level,
                                                            unsigned int index,
                                              unsigned int orth_basis_index, enumf value)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    assert(orth_basis_index < dimensions);
    center_partsum[tree_level * dimensions * max_nodes_per_level +
                   orth_basis_index * max_nodes_per_level + index] = value;
  }

  __device__ __host__ inline enumf
  get_center_partsum(unsigned int tree_level, unsigned int index,
                                               unsigned int orth_basis_index)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    assert(orth_basis_index < dimensions);
    return center_partsum[tree_level * dimensions * max_nodes_per_level +
                          orth_basis_index * max_nodes_per_level + index];
  }

  __device__ __host__ inline enumi get_coefficient(unsigned int tree_level, unsigned int index,
                                            unsigned int coordinate)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    assert(coordinate < dimensions_per_level);
    return coefficients[tree_level * dimensions_per_level * max_nodes_per_level +
                        coordinate * max_nodes_per_level + index];
  }

  __device__ __host__ inline void
  set_coefficient(unsigned int tree_level, unsigned int index,
                                           unsigned int coordinate, enumi value)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    assert(coordinate < dimensions_per_level);
    coefficients[tree_level * dimensions_per_level * max_nodes_per_level +
                 coordinate * max_nodes_per_level + index] = value;
  }

  __device__ __host__ inline unsigned int get_parent_index(unsigned int tree_level,
                                                                  unsigned int index)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    return parent_indices[tree_level * max_nodes_per_level + index];
  }

  __device__ __host__ inline enumf get_partdist(unsigned int tree_level, unsigned int index)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    return partdist[tree_level * max_nodes_per_level + index];
  }

  __device__ __host__ inline void set_partdist(unsigned int tree_level, unsigned int index,
                                                      enumf value)
  {
    assert(tree_level < levels);
    assert(index < max_nodes_per_level);
    partdist[tree_level * max_nodes_per_level + index] = value;
  }

  __device__ __host__ inline unsigned int get_node_count(unsigned int tree_level)
  {
    assert(tree_level < levels);
    return open_node_count[tree_level];
  }

  __device__ __host__ inline unsigned int add_subtree(unsigned int tree_level,
                                                      unsigned int parent_node_index)
  {
    assert(tree_level < levels);
    const unsigned int new_task_index = aggregated_atomic_inc(&open_node_count[tree_level]);
    assert(new_task_index < max_nodes_per_level);
    parent_indices[tree_level * max_nodes_per_level + new_task_index] = parent_node_index;
    return new_task_index;
  }

  template <typename CG, unsigned int block_size>
  __device__ __host__ inline void
  filter_nodes(CG &cooperative_group, PrefixCounter<CG, block_size> &prefix_counter,
               unsigned int tree_level, unsigned int old_index, bool keep_this_thread_task,
               unsigned int active_thread_count)
  {
    assert(tree_level < levels);
    assert(active_thread_count <= open_node_count[tree_level]);
    assert(old_index ==
           open_node_count[tree_level] - active_thread_count + cooperative_group.thread_rank());
    assert(tree_level + 1 == levels || open_node_count[tree_level + 1] == 0);

    unsigned int kept_tasks = 0;
    const bool is_active =
        keep_this_thread_task && cooperative_group.thread_rank() < active_thread_count;
    const unsigned int new_offset =
        prefix_counter.prefix_count(cooperative_group, is_active, kept_tasks);
    const unsigned int new_index = new_offset + open_node_count[tree_level] - active_thread_count;

    enumi coefficients_tmp[dimensions_per_level];
    enumf center_partsum_tmp[dimensions];
    enumf partdist_tmp;
    unsigned int parent_index_tmp;
    if (is_active)
    {
      partdist_tmp     = partdist[tree_level * max_nodes_per_level + old_index];
      parent_index_tmp = parent_indices[tree_level * max_nodes_per_level + old_index];
      for (unsigned int i = 0; i < dimensions_per_level; ++i)
      {
        coefficients_tmp[i] =
            enumeration_x[tree_level * dimensions_per_level * max_nodes_per_level +
                          i * max_nodes_per_level + old_index];
      }
      for (unsigned int i = 0; i < dimensions; ++i)
      {
        center_partsum_tmp[i] = center_partsum[tree_level * dimensions * max_nodes_per_level +
                                               i * max_nodes_per_level + old_index];
      }
    }

    cooperative_group.sync();

    if (is_active)
    {
      partdist[tree_level * max_nodes_per_level + new_index]       = partdist_tmp;
      parent_indices[tree_level * max_nodes_per_level + new_index] = parent_index_tmp;
      for (unsigned int i = 0; i < dimensions_per_level; ++i)
      {
        enumeration_x[tree_level * dimensions_per_level * max_nodes_per_level +
                      i * max_nodes_per_level + new_index] = coefficients_tmp[i];
      }
      for (unsigned int i = 0; i < dimensions; ++i)
      {
        center_partsum[tree_level * dimensions * max_nodes_per_level + i * max_nodes_per_level +
                       new_index] = center_partsum_tmp[i];
      }
    }

    if (cooperative_group.thread_rank() == 0)
    {
      open_node_count[tree_level] -= active_thread_count - kept_tasks;
    }
  }
};

template <unsigned int levels, unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
struct ProcessLeafCallback
{
  unsigned int level;
  unsigned int parent_index;
  Matrix mu;
  uint32_t *radius_squared_location;
  SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer;

  __device__ __host__ void operator()(const enumi *x, enumf squared_norm);
};

template <unsigned int levels, unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
__device__ __host__ inline void
ProcessLeafCallback<levels, dimensions_per_level, max_nodes_per_level>::operator()(
    const enumi *x, enumf squared_norm)
{
  if (squared_norm == 0)
  {
    return;
  }

  uint32_t squared_norm_repr = float_to_int_order_preserving_bijection(squared_norm);
  uint32_t old_repr          = atomic_min(radius_squared_location, squared_norm_repr);

  if (old_repr > squared_norm_repr)
  {
    // Here save the found result
    if (TRACE)
    {
      printf("Squared norm %f: ", squared_norm);
      float coefficient;
      for (unsigned int i = 0; i < dimensions_per_level; ++i)
      {
        coefficient = x[i];
        printf("%f, ", coefficient);
      }
      unsigned int index = parent_index;
      for (int j = levels - 1; j > 0; --j)
      {
        for (unsigned int i = 0; i < dimensions_per_level; ++i)
        {
          coefficient = buffer.get_coefficient(j, index, i);
          printf("%f, ", coefficient);
        }
        index = buffer.get_parent_index(j, index);
      }
      printf("; Start point index: %d", buffer.get_parent_index(0, index));
      printf("\n");
    }
  }
}

template <unsigned int levels, unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
struct AddToTreeCallback
{
  unsigned int level;
  unsigned int parent_index;
  Matrix mu;
  SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer;
  PerfCounter &counter;

  __device__ __host__ void operator()(const enumi *x, enumf squared_norm);
};

template <unsigned int levels, unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
__device__ __host__ inline void
AddToTreeCallback<levels, dimensions_per_level, max_nodes_per_level>::operator()(const enumi *x,
                                                                                 enumf squared_norm)
{
  assert(level > 0);

  const unsigned int new_index = buffer.add_subtree(level, parent_index);
  for (unsigned int j = 0; j < dimensions_per_level; ++j)
  {
    buffer.set_coefficient(level, new_index, j, x[j]);
  }
  buffer.set_partdist(level, new_index, squared_norm);
  // subtree initialization will be done later in a synchronized way
}

template <unsigned int levels, unsigned int dimensions_per_level>
__device__ __host__ inline enumf calc_center_partsum(
    unsigned int level, unsigned int index, unsigned int center_partsum_index,
    enumi x[dimensions_per_level], Matrix mu)
{
  unsigned int kk_offset = (levels - level - 1) * dimensions_per_level;
  enumf center_partsum   = 0;
  for (unsigned int j = 0; j < dimensions_per_level; ++j)
  {
    center_partsum -= x[j] * mu.at(center_partsum_index, j + dimensions_per_level + kk_offset);
  }
  assert(!isnan(center_partsum));
  return center_partsum;
}

template <typename CG, unsigned int levels, unsigned int dimensions_per_level,
          unsigned int max_nodes_per_level>
__device__ __host__ inline void calc_center_partsums(
    CG &group, unsigned int level, unsigned int already_calculated_node_count,
    SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer, Matrix mu,
    PerfCounter &counter)
{
  for (unsigned int new_index = already_calculated_node_count + group.thread_rank();
       new_index < buffer.get_node_count(level); new_index += group.size())
  {
    unsigned int kk_offset = (levels - level - 1) * dimensions_per_level;
    unsigned int center_i  = kk_offset + dimensions_per_level - 1;

    const unsigned int parent_index = buffer.get_parent_index(level, new_index);
    enumi x[dimensions_per_level];
    for (unsigned int j = 0; j < dimensions_per_level; ++j)
    {
      x[j] = buffer.get_coefficient(level, new_index, j);
    }
    
    unsigned int i = 0;
    enumf center_partsum;
    enumf preloaded_parent_center_partsums[4];
    preloaded_parent_center_partsums[0] = buffer.get_center_partsum(level - 1, parent_index, 0);
    preloaded_parent_center_partsums[1] = buffer.get_center_partsum(level - 1, parent_index, 1);
    preloaded_parent_center_partsums[2] = buffer.get_center_partsum(level - 1, parent_index, 2);
    for (; i + 6 < kk_offset + dimensions_per_level; i += 4)
    {
      preloaded_parent_center_partsums[3] = buffer.get_center_partsum(level - 1, parent_index, i + 3);
      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i, x, mu);
      buffer.set_center_partsum(level, new_index, i, center_partsum);

      preloaded_parent_center_partsums[0] =
          buffer.get_center_partsum(level - 1, parent_index, i + 4);
      center_partsum =
          preloaded_parent_center_partsums[1] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 1, x, mu);
      buffer.set_center_partsum(level, new_index, i + 1, center_partsum);

      preloaded_parent_center_partsums[1] =
          buffer.get_center_partsum(level - 1, parent_index, i + 5);
      center_partsum =
          preloaded_parent_center_partsums[2] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 2, x, mu);
      buffer.set_center_partsum(level, new_index, i + 2, center_partsum);

      preloaded_parent_center_partsums[2] =
          buffer.get_center_partsum(level - 1, parent_index, i + 6);
      center_partsum =
          preloaded_parent_center_partsums[3] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 3, x, mu);
      buffer.set_center_partsum(level, new_index, i + 3, center_partsum);
    }
    if (i + 6 == kk_offset + dimensions_per_level)
    {
      preloaded_parent_center_partsums[3] =
          buffer.get_center_partsum(level - 1, parent_index, i + 3);

      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i, x, mu);
      buffer.set_center_partsum(level, new_index, i, center_partsum);

      preloaded_parent_center_partsums[0] =
          buffer.get_center_partsum(level - 1, parent_index, i + 4);

      center_partsum =
          preloaded_parent_center_partsums[1] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 1, x, mu);
      buffer.set_center_partsum(level, new_index, i + 1, center_partsum);

      preloaded_parent_center_partsums[1] =
          buffer.get_center_partsum(level - 1, parent_index, i + 5);

      center_partsum =
          preloaded_parent_center_partsums[2] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 2, x, mu);
      buffer.set_center_partsum(level, new_index, i + 2, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[3] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 3, x, mu);
      buffer.set_center_partsum(level, new_index, i + 3, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 4, x, mu);
      buffer.set_center_partsum(level, new_index, i + 4, center_partsum);
      
      center_partsum =
          preloaded_parent_center_partsums[1] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 5, x, mu);
      buffer.set_center_partsum(level, new_index, i + 5, center_partsum);
    }
    if (i + 5 == kk_offset + dimensions_per_level)
    {
      preloaded_parent_center_partsums[3] =
          buffer.get_center_partsum(level - 1, parent_index, i + 3);

      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i, x, mu);
      buffer.set_center_partsum(level, new_index, i, center_partsum);

      preloaded_parent_center_partsums[0] =
          buffer.get_center_partsum(level - 1, parent_index, i + 4);

      center_partsum =
          preloaded_parent_center_partsums[1] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 1, x, mu);
      buffer.set_center_partsum(level, new_index, i + 1, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[2] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 2, x, mu);
      buffer.set_center_partsum(level, new_index, i + 2, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[3] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 3, x, mu);
      buffer.set_center_partsum(level, new_index, i + 3, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 4, x, mu);
      buffer.set_center_partsum(level, new_index, i + 4, center_partsum);
    }
    if (i + 4 == kk_offset + dimensions_per_level)
    {
      preloaded_parent_center_partsums[3] =
          buffer.get_center_partsum(level - 1, parent_index, i + 3);

      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i, x, mu);
      buffer.set_center_partsum(level, new_index, i, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[1] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 1, x, mu);
      buffer.set_center_partsum(level, new_index, i + 1, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[2] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 2, x, mu);
      buffer.set_center_partsum(level, new_index, i + 2, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[3] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 3, x, mu);
      buffer.set_center_partsum(level, new_index, i + 3, center_partsum);
    }
    if (i + 3 == kk_offset + dimensions_per_level)
    {
      center_partsum =
          preloaded_parent_center_partsums[0] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i, x, mu);
      buffer.set_center_partsum(level, new_index, i, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[1] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 1, x, mu);
      buffer.set_center_partsum(level, new_index, i + 1, center_partsum);

      center_partsum =
          preloaded_parent_center_partsums[2] +
          calc_center_partsum<levels, dimensions_per_level>(level, new_index, i + 2, x, mu);
      buffer.set_center_partsum(level, new_index, i + 2, center_partsum);
    }

    enumf center = buffer.get_center_partsum(level, new_index, center_i);
    assert(!isnan(center));
    buffer.set_center_partsum(level, new_index, center_i, center);

    enumf partdist = buffer.get_partdist(level, new_index);
    buffer.init_subtree(level, new_index, partdist, center);
  }

  unsigned long long end = time();
}

// needs synchronization with operations that modify tree_level level - 1 of the buffer
template <typename CG, unsigned int levels, unsigned int dimensions_per_level,
          unsigned int max_nodes_per_level>
__device__ __host__ void
do_search_step(CG &group, SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer,
               unsigned int level, Matrix mu, const enumf *rdiag, uint32_t *radius_squared_location,
               unsigned int max_subtree_paths, PerfCounter &counter)
{
  unsigned long long begin = time();

  const unsigned int active_thread_count = min(buffer.get_node_count(level), group.size());
  const unsigned int index =
      buffer.get_node_count(level) - active_thread_count + group.thread_rank();
  const bool active = index < buffer.get_node_count(level);

  const unsigned int offset_kk = (levels - level - 1) * dimensions_per_level;
  unsigned int max_paths = max_subtree_paths;

  if (level < levels - 1)
  {
    unsigned int existing_nodes = buffer.get_node_count(level + 1);
    group.sync();
    if (active)
    {
      CudaEnumeration<dimensions_per_level> enumeration = buffer.get_enumeration(
          level, index, mu.block(offset_kk, offset_kk), rdiag, radius_squared_location);

      typedef AddToTreeCallback<levels, dimensions_per_level, max_nodes_per_level> CallbackType;
      CallbackType callback = {level + 1, index, mu, buffer, counter};
      enumeration.template enumerate_recursive<dimensions_per_level - 1, CallbackType>(
          callback, max_paths, counter);

      buffer.set_enumeration(level, index, enumeration);
    }
    group.sync();
    calc_center_partsums(group, level + 1, existing_nodes, buffer, mu, counter);
  }
  else
  {
    if (active)
    {
      CudaEnumeration<dimensions_per_level> enumeration = buffer.get_enumeration(
          level, index, mu.block(offset_kk, offset_kk), rdiag, radius_squared_location);

      typedef ProcessLeafCallback<levels, dimensions_per_level, max_nodes_per_level> CallbackT;
      CallbackT callback = {level + 1, index, mu, radius_squared_location, buffer};
      enumeration.template enumerate_recursive<dimensions_per_level - 1, CallbackT>(
          callback, max_paths, counter);

      buffer.set_enumeration(level, index, enumeration);
    }
  }

  unsigned long long end = time();
  counter.perf_count(2, end - begin);
}

template <typename CG, unsigned int levels, unsigned int dimensions_per_level,
          unsigned int max_nodes_per_level>
__device__ __host__ inline void get_done_subtree_count(
    CG &group, unsigned int *shared_counter,
    SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer,
    unsigned int level, Matrix mu, const enumf *rdiag, const uint32_t *radius_square_location)
{
  const unsigned int active_thread_count = min(buffer.get_node_count(level), group.size());
  const unsigned int index =
      buffer.get_node_count(level) - active_thread_count + group.thread_rank();
  const bool active = index < buffer.get_node_count(level);

  const unsigned int offset_kk = (levels - level - 1) * dimensions_per_level;

  if (active)
  {
    bool is_done = buffer
                       .get_enumeration(level, index, mu.block(offset_kk, offset_kk), rdiag,
                                        radius_square_location)
                       .template is_enumeration_done<dimensions_per_level - 1>();
    if (is_done)
    {
      aggregated_atomic_inc(shared_counter);
    }
  }
}

// needs synchronization with operations that work on tree_level level
template <typename CG, unsigned int block_size, unsigned int levels,
          unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
__device__ __host__ inline void
do_cleanup_step(CG &group, PrefixCounter<CG, block_size> &prefix_counter,
                SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer,
                unsigned int level, Matrix mu, const enumf *rdiag,
                const uint32_t *radius_square_location)
{
  const unsigned int active_thread_count = min(buffer.get_node_count(level), group.size());
  const unsigned int index =
      buffer.get_node_count(level) - active_thread_count + group.thread_rank();
  const bool active = index < buffer.get_node_count(level);

  const unsigned int offset_kk = (levels - level - 1) * dimensions_per_level;

  bool is_done = buffer
                     .get_enumeration(level, index, mu.block(offset_kk, offset_kk), rdiag,
                                      radius_square_location)
                     .template is_enumeration_done<dimensions_per_level - 1>();

  group.sync();

  buffer.filter_nodes(group, prefix_counter, level, index, !is_done, active_thread_count);
}

template <typename CG, unsigned int block_size, unsigned int levels,
          unsigned int dimensions_per_level, unsigned int max_nodes_per_level>
__device__ __host__ inline void
clear_level(CG &group, PrefixCounter<CG, block_size> &prefix_counter,
            unsigned int *shared_counter,
            SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> &buffer,
            int level, Matrix mu, const enumf *rdiag,
            uint32_t *radius_square_location,
            unsigned int max_subtree_paths, PerfCounter counter)
{
  unsigned long long begin = time();
  while (level >= 0)
  {
    assert(all_threads_eq(group, level, shared_counter));
    if (level + 1 < levels)
    {
      if (buffer.get_node_count(level) > 0)
      {
        const unsigned int active_thread_count = min(buffer.get_node_count(level), group.size());
        // create as many children as fit into the children buffer (and only as long as we can generate using enough parallel threads)
        while (true)
        {
          do_search_step(group, buffer, level, mu, rdiag, radius_square_location,
                         max_subtree_paths, counter);
          if (group.thread_rank() == 0)
          {
            *shared_counter = 0;
          }
          group.sync();
          get_done_subtree_count(group, shared_counter, buffer, level, mu, rdiag,
                                 radius_square_location);
          group.sync();
          
          debug_message_thread(
                "Worked on level %d, next level points are %d, %d nodes of current working pool "
                   "(%d) are done\n",
                   level, buffer.get_node_count(level + 1), *shared_counter, active_thread_count);

          if (buffer.get_node_count(level + 1) >= max_nodes_per_level / 2)
          {
            break;
          }
          else if (*shared_counter >= active_thread_count / 2)
          {
            break;
          }
          group.sync();
        }
        ++level;
      }
      else
      {
        --level;
        if (level >= 0)
        {
          group.sync();
          do_cleanup_step(group, prefix_counter, buffer, level, mu, rdiag,
                          radius_square_location);
          group.sync();

          debug_message_thread("Cleaned up level %d, has now %d nodes\n",
                   level, buffer.get_node_count(level));

          group.sync();
        }
      }
    }
    else
    {
      while (buffer.get_node_count(level) > 0)
      {
        do_search_step(group, buffer, level, mu, rdiag, radius_square_location,
                       max_subtree_paths, counter);
        do_cleanup_step(group, prefix_counter, buffer, level, mu, rdiag,
                        radius_square_location);
        group.sync();
      }
      --level;
      if (level >= 0)
      {
        group.sync();
        do_cleanup_step(group, prefix_counter, buffer, level, mu, rdiag,
                        radius_square_location);
        group.sync();
      }
    }
  }
  unsigned long long end = time();
  counter.perf_count(3, end - begin);
}

constexpr __device__ __host__ unsigned int constexpr_max(unsigned int a, unsigned int b)
{
  if (a > b)
  {
    return a;
  }
  else
  {
    return b;
  }
}

constexpr unsigned int search_block_size = 128;

template <unsigned int levels, unsigned int dimensions_per_level,
          unsigned int max_nodes_per_level>
__global__ void __launch_bounds__(search_block_size, 1)
    search_kernel(unsigned char *buffer_memory, const enumi *start_points,
                  unsigned int *processed_start_point_counter, unsigned int start_point_dim,
                  unsigned int start_point_count, const enumf *mu_ptr, const enumf *rdiag,
                  uint32_t *radius_squared_location, unsigned int max_subtree_paths,
                  unsigned long long *perf_counter_memory, const unsigned int nodes_per_group)
{
  unsigned long long begin = time();

  typedef cooperative_groups::thread_block_tile<32> CG;
  typedef SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> SubtreeBuffer;

  constexpr unsigned int block_size = search_block_size;
  constexpr unsigned int dimensions = dimensions_per_level * levels;
  constexpr unsigned int group_count_per_block = block_size / 32;

  constexpr unsigned int mu_shared_memory_size = dimensions * dimensions * sizeof(enumf);

  constexpr unsigned int shared_mem_size =
      group_count_per_block * sizeof(unsigned int) + mu_shared_memory_size;

  extern __shared__ unsigned char shared_mem[shared_mem_size];

  CG group = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());
  const unsigned int group_id          = thread_id() / 32;
  const unsigned int group_id_in_block = thread_id_in_block() / 32;

  PrefixCounter<CG, block_size> prefix_counter;

  unsigned int *group_shared_counter =
      reinterpret_cast<unsigned int *>(shared_mem + group_id_in_block * sizeof(unsigned int));

  enumf *mu_shared =
      reinterpret_cast<enumf *>(shared_mem + group_count_per_block * sizeof(unsigned int));

  const unsigned int ldmu = dimensions + start_point_dim;
  for (unsigned int i = threadIdx.x; i < dimensions * dimensions; i += blockDim.x)
  {
    mu_shared[i] = mu_ptr[i / dimensions * ldmu + i % dimensions];
  }
  __syncthreads();
  Matrix mu(mu_shared, dimensions);

  PerfCounter counter(perf_counter_memory);

  assert(nodes_per_group <= group.size());

  SubtreeBuffer buffer(buffer_memory + group_id * SubtreeBuffer::memory_size_in_bytes);

  while (true)
  {
    group.sync();
    if (group.thread_rank() == 0)
    {
      *group_shared_counter = atomic_add(processed_start_point_counter, nodes_per_group);
    }
    buffer.init(group);
    group.sync();

    if (*group_shared_counter >= start_point_count)
    {
      break;
    }
    const unsigned int start_point_index = *group_shared_counter + group.thread_rank();
    const bool active =
        group.thread_rank() < nodes_per_group && start_point_index < start_point_count;

    if (active)
    {
      const enumi *start_point = &start_points[start_point_index * start_point_dim];
      const unsigned int index = buffer.add_subtree(0, start_point_index);
      for (unsigned int i = 0; i < dimensions; ++i)
      {
        enumf center_partsum = 0;
        for (unsigned int j = 0; j < start_point_dim; ++j)
        {
          center_partsum -= start_point[j] * mu_ptr[i * ldmu + dimensions + j];
        }
        buffer.set_center_partsum(0, index, i, center_partsum);
      }
      enumf partdist = 0;
      for (int j = 0; j < start_point_dim; ++j)
      {
        enumf alpha = start_point[j];
        for (unsigned int i = j + 1; i < start_point_dim; ++i)
        {
          alpha += start_point[i] * mu_ptr[(j + dimensions) * ldmu + i + dimensions];
        }
        partdist += alpha * alpha * rdiag[dimensions + j];
      }
      buffer.init_subtree(0, index, partdist, buffer.get_center_partsum(0, index, dimensions - 1));
    }
    debug_message_thread("Get %d new nodes\n", nodes_per_group);

    group.sync();

    clear_level<CG, block_size, levels, dimensions_per_level, max_nodes_per_level>(
        group, prefix_counter, group_shared_counter, buffer, 0, mu, rdiag,
        radius_squared_location, max_subtree_paths, counter);
  }

  unsigned long long end = time();
  counter.perf_count(4, end - begin);
}

template <unsigned int levels, unsigned int dimensions_per_level,
          unsigned int max_nodes_per_level, unsigned int start_point_dim>
void search(const std::array<std::array<float, levels * dimensions_per_level + start_point_dim>,
                             levels * dimensions_per_level + start_point_dim> &mu,
            const std::vector<std::array<enumi, start_point_dim>> &start_points,
            unsigned int max_subtree_paths, unsigned int grid_size, unsigned int nodes_per_group)
{
  typedef SubtreeEnumerationBuffer<levels, dimensions_per_level, max_nodes_per_level> SubtreeBuffer;

  constexpr unsigned int dimensions = dimensions_per_level * levels;
  constexpr unsigned int mu_n       = dimensions + start_point_dim;

  const unsigned int group_count = grid_size * search_block_size / 32;
  const unsigned int group_size  = 32;
  assert(max_nodes_per_level >= max_subtree_paths * group_size);

  CudaPtr<unsigned char> buffer_mem =
      alloc(unsigned char, SubtreeBuffer::memory_size_in_bytes *group_count);
  CudaPtr<uint32_t> radius_mem        = alloc(uint32_t, 1);
  CudaPtr<enumf> device_mu            = alloc(enumf, mu_n * mu_n);
  CudaPtr<enumf> device_rdiag         = alloc(enumf, mu_n);
  CudaPtr<unsigned long long> counter = alloc(unsigned long long, 5);
  CudaPtr<enumi> device_start_points  = alloc(enumi, start_points.size() * start_point_dim);
  CudaPtr<unsigned int> processed_start_point_count = alloc(unsigned int, 1);

  const enumf radius                     = find_initial_radius(mu) * 1.01;
  const uint32_t radius_squared_location = float_to_int_order_preserving_bijection(radius * radius);
  std::unique_ptr<float[]> host_mu(new float[mu_n * mu_n]);
  std::unique_ptr<float[]> host_rdiag(new float[mu_n]);
  for (unsigned int i = 0; i < mu_n; ++i)
  {
    host_rdiag[i] = mu[i][i];
    for (unsigned int j = 0; j < mu_n; ++j)
    {
      host_mu[i * mu_n + j] = mu[i][j] / host_rdiag[i];
    }
    host_rdiag[i] = host_rdiag[i] * host_rdiag[i];
  }

  check(hipMemcpy(device_mu.get(), host_mu.get(), mu_n * mu_n * sizeof(enumf),
                   hipMemcpyHostToDevice));
  check(hipMemcpy(device_rdiag.get(), host_rdiag.get(), mu_n * sizeof(enumf),
                   hipMemcpyHostToDevice));
  check(hipMemcpy(radius_mem.get(), &radius_squared_location, sizeof(uint32_t),
                   hipMemcpyHostToDevice));
  check(hipMemcpy(device_start_points.get(), start_points[0].data(),
                   start_point_dim * start_points.size() * sizeof(enumi), hipMemcpyHostToDevice));

  std::cout << "started " << grid_size << " block with " << search_block_size << " threads each"
            << std::endl;

  std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();
  search_kernel<levels, dimensions_per_level, max_nodes_per_level>
      <<<dim3(grid_size), dim3(search_block_size)>>>(
          buffer_mem.get(), device_start_points.get(), processed_start_point_count.get(),
          start_point_dim, start_points.size(), device_mu.get(), device_rdiag.get(),
          radius_mem.get(), max_subtree_paths, counter.get(), nodes_per_group);

  check(hipDeviceSynchronize());
  check(hipGetLastError());
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout << "time: "
            << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms"
            << std::endl;

  unsigned long long searched_nodes;
  uint32_t result_radius;
  check(hipMemcpy(&searched_nodes, counter.get(), sizeof(unsigned long long),
                   hipMemcpyDeviceToHost));
  check(hipMemcpy(&result_radius, radius_mem.get(), sizeof(uint32_t), hipMemcpyDeviceToHost));
  std::cout << "searched nodes: " << searched_nodes << std::endl;
  std::cout << "result radius: " << sqrt(int_to_float_order_preserving_bijection(result_radius))
            << std::endl;
  print_performance_counter(&counter.get()[1]);
  print_performance_counter(&counter.get()[2]);
  print_performance_counter(&counter.get()[3]);
  print_performance_counter(&counter.get()[4]);
}
